#include "hip/hip_runtime.h"
#include "./cuda_ops.cuh"

template <typename FloatType>
struct BlockPrefixCallbackOp
{
    // Running prefix
    FloatType running_total;
    // Constructor
    __device__ BlockPrefixCallbackOp(FloatType running_total) : running_total(running_total) {}
    // Callback operator to be entered by the first warp of threads in the block.
    // Thread-0 is responsible for returning a value for seeding the block-wide scan.
    __device__ FloatType operator()(FloatType block_aggregate)
    {
        FloatType old_prefix = running_total;
        running_total += block_aggregate;
        return old_prefix;
    }
};

template <typename IdType, typename FloatType, int TILE_SIZE, int BLOCK_WARPS, int WARP_SIZE>
__global__ void _CSRRowWiseSampleReplaceKernel(
    const uint64_t rand_seed,
    const int64_t num_picks,
    const int64_t num_rows,
    const IdType *const in_rows,
    const IdType *const in_ptr,
    const IdType *const in_cols,
    const FloatType *const prob,
    const IdType *const out_ptr,
    const IdType *const cdf_ptr,
    FloatType *const cdf,
    IdType *const out_rows,
    IdType *const out_cols)
{
    // we assign one warp per row
    assert(blockDim.x == WARP_SIZE);
    assert(blockDim.y == BLOCK_WARPS);

    int64_t out_row = blockIdx.x * TILE_SIZE + threadIdx.y;
    const int64_t last_row = min(static_cast<int64_t>(blockIdx.x + 1) * TILE_SIZE, num_rows);

    hiprandStatePhilox4_32_10_t rng;
    hiprand_init(rand_seed * gridDim.x + blockIdx.x, threadIdx.y * BLOCK_WARPS + threadIdx.x, 0, &rng);

    typedef hipcub::WarpScan<FloatType> WarpScan;
    __shared__ typename WarpScan::TempStorage temp_storage[BLOCK_WARPS];
    int warp_id = threadIdx.y;
    int laneid = threadIdx.x;

    while (out_row < last_row)
    {
        const int64_t row = in_rows[out_row];
        const int64_t in_row_start = in_ptr[row];
        const int64_t out_row_start = out_ptr[out_row];
        const int64_t cdf_row_start = cdf_ptr[out_row];
        const int64_t deg = in_ptr[row + 1] - in_row_start;
        const FloatType MIN_THREAD_DATA = static_cast<FloatType>(0.0f);

        if (deg > 0)
        {
            int64_t max_iter = (1 + (deg - 1) / WARP_SIZE) * WARP_SIZE;
            // Have the block iterate over segments of items

            FloatType warp_aggregate = static_cast<FloatType>(0.0f);
            for (int64_t idx = laneid; idx < max_iter; idx += WARP_SIZE)
            {
                FloatType thread_data = idx < deg ? prob[in_row_start + idx] : MIN_THREAD_DATA;
                if (laneid == 0)
                    thread_data += warp_aggregate;
                thread_data = max(thread_data, MIN_THREAD_DATA);

                WarpScan(temp_storage[warp_id]).InclusiveSum(thread_data, thread_data, warp_aggregate);
                __syncwarp();
                // Store scanned items to cdf array
                if (idx < deg)
                {
                    cdf[cdf_row_start + idx] = thread_data;
                }
            }
            __syncwarp();

            for (int64_t idx = laneid; idx < num_picks; idx += WARP_SIZE)
            {
                // get random value
                FloatType sum = cdf[cdf_row_start + deg - 1];
                FloatType rand = static_cast<FloatType>(hiprand_uniform(&rng) * sum);
                // get the offset of the first value within cdf array which is greater than random value.
                int64_t item = hipcub::UpperBound<FloatType *, int64_t, FloatType>(
                    &cdf[cdf_row_start], deg, rand);
                item = min(item, deg - 1);
                // get in and out index
                const int64_t in_idx = in_row_start + item;
                const int64_t out_idx = out_row_start + idx;
                // copy permutation over
                out_rows[out_idx] = static_cast<IdType>(row);
                out_cols[out_idx] = in_cols[in_idx];
            }
        }
        out_row += BLOCK_WARPS;
    }
}

std::vector<torch::Tensor> RowWiseSamplingProb_CDF(
    torch::Tensor seeds,
    torch::Tensor indptr,
    torch::Tensor indices,
    torch::Tensor probs,
    int64_t num_picks,
    bool replace)
{
    int num_rows = seeds.numel();
    torch::Tensor sub_indptr, temp_indptr;
    std::tie(sub_indptr, temp_indptr) = _GetSubAndTempIndptr<int64_t>(seeds, indptr, num_picks, replace);
    thrust::device_ptr<int64_t> sub_prefix(static_cast<int64_t *>(sub_indptr.data_ptr<int64_t>()));
    thrust::device_ptr<int64_t> temp_prefix(static_cast<int64_t *>(temp_indptr.data_ptr<int64_t>()));
    int nnz = sub_prefix[num_rows];
    int temp_size = temp_prefix[num_rows];

    torch::Tensor coo_row = torch::empty(nnz, seeds.options());
    torch::Tensor coo_col = torch::empty(nnz, indices.options());
    torch::Tensor temp = torch::empty(temp_size, probs.options());

    const uint64_t random_seed = 7777;
    constexpr int WARP_SIZE = 32;
    constexpr int BLOCK_WARPS = BLOCK_SIZE / WARP_SIZE;
    constexpr int TILE_SIZE = 16;
    if (replace)
    {
        const dim3 block(WARP_SIZE, BLOCK_WARPS);
        const dim3 grid((num_rows + TILE_SIZE - 1) / TILE_SIZE);

        _CSRRowWiseSampleReplaceKernel<int64_t, float, TILE_SIZE, BLOCK_WARPS, WARP_SIZE><<<grid, block>>>(
            random_seed,
            num_picks,
            num_rows,
            seeds.data_ptr<int64_t>(),
            indptr.data_ptr<int64_t>(),
            indices.data_ptr<int64_t>(),
            probs.data_ptr<float>(),
            sub_indptr.data_ptr<int64_t>(),
            temp_indptr.data_ptr<int64_t>(),
            temp.data_ptr<float>(),
            coo_row.data_ptr<int64_t>(),
            coo_col.data_ptr<int64_t>());
    }
    else
    {
        printf("Not Implemented.\n");
    }

    return {coo_row, coo_col};
}

static auto registry =
    torch::RegisterOperators(
        "gswp::RowWiseSamplingProb_CDF(Tensor seeds, Tensor indptr, Tensor indices, Tensor probs, int num_pick, bool replace) -> Tensor[]",
        &RowWiseSamplingProb_CDF);